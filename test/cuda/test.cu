
#include <hip/hip_runtime.h>
extern "C" {

    __global__ void inc (int n, float* a) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) {
            a[i] += 1;
        }
    };

    __device__ float gpu_a[] = {1.0, 2.0, 3.0};

}
